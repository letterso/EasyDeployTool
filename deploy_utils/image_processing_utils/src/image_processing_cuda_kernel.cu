
#include <hip/hip_runtime.h>

extern "C" {

__device__ int clamp(int x, int low, int high)
{
  return x < low ? low : (x > high ? high : x);
}

__global__ void ResizePadNormPadvalueKernel(const unsigned char *src,
                                            int                  src_h,
                                            int                  src_w,
                                            int                  src_stride,
                                            int                  src_format_bgr,
                                            float               *dst,
                                            int                  dst_h,
                                            int                  dst_w,
                                            int                  pad_top,
                                            int                  pad_left,
                                            float                scale,
                                            float                mean0,
                                            float                mean1,
                                            float                mean2,
                                            float                val0,
                                            float                val1,
                                            float                val2,
                                            bool                 do_transpose,
                                            bool                 do_norm,
                                            int                  pad_value,
                                            float                pad_color0,
                                            float                pad_color1,
                                            float                pad_color2)
{
  int dx = blockIdx.x * blockDim.x + threadIdx.x;
  int dy = blockIdx.y * blockDim.y + threadIdx.y;
  if (dx >= dst_w || dy >= dst_h)
    return;

  float out_c[3] = {0};

  int  x        = dx - pad_left;
  int  y        = dy - pad_top;
  bool in_range = (x >= 0 && x < int(src_w * scale) && y >= 0 && y < int(src_h * scale));

  if (in_range)
  {
    float src_x                = x / scale;
    float src_y                = y / scale;
    int   isrc_x               = static_cast<int>(roundf(src_x));
    int   isrc_y               = static_cast<int>(roundf(src_y));
    isrc_x                     = clamp(isrc_x, 0, src_w - 1);
    isrc_y                     = clamp(isrc_y, 0, src_h - 1);
    const unsigned char *p     = src + isrc_y * src_stride + isrc_x * 3;
    int                  r_idx = src_format_bgr ? 2 : 0;
    int                  g_idx = 1;
    int                  b_idx = src_format_bgr ? 0 : 2;
    out_c[0]                   = static_cast<float>(p[r_idx]);
    out_c[1]                   = static_cast<float>(p[g_idx]);
    out_c[2]                   = static_cast<float>(p[b_idx]);
  } else
  {
    // -------- pad区，按pad value 逻辑处理 -------------------
    if (pad_value == 0)
    { // EDGE
      int                  src_x = clamp((int)roundf((x / scale)), 0, src_w - 1);
      int                  src_y = clamp((int)roundf((y / scale)), 0, src_h - 1);
      const unsigned char *p     = src + src_y * src_stride + src_x * 3;
      int                  r_idx = src_format_bgr ? 2 : 0;
      int                  g_idx = 1;
      int                  b_idx = src_format_bgr ? 0 : 2;
      out_c[0]                   = static_cast<float>(p[r_idx]);
      out_c[1]                   = static_cast<float>(p[g_idx]);
      out_c[2]                   = static_cast<float>(p[b_idx]);
    } else
    { // CONSTANT
      out_c[0] = pad_color0;
      out_c[1] = pad_color1;
      out_c[2] = pad_color2;
    }
  }

  if (do_norm)
  {
    out_c[0] = (out_c[0] - mean0) / val0;
    out_c[1] = (out_c[1] - mean1) / val1;
    out_c[2] = (out_c[2] - mean2) / val2;
  }

  int single_channel = dst_h * dst_w;
  if (do_transpose)
  {
    dst[0 * single_channel + dy * dst_w + dx] = out_c[0];
    dst[1 * single_channel + dy * dst_w + dx] = out_c[1];
    dst[2 * single_channel + dy * dst_w + dx] = out_c[2];
  } else
  {
    int idx      = (dy * dst_w + dx) * 3;
    dst[idx + 0] = out_c[0];
    dst[idx + 1] = out_c[1];
    dst[idx + 2] = out_c[2];
  }
}

void launch_resize_pad_norm(const unsigned char *src,
                            int                  src_h,
                            int                  src_w,
                            int                  src_stride,
                            int                  src_format_bgr,
                            float               *dst,
                            int                  dst_h,
                            int                  dst_w,
                            int                  pad_top,
                            int                  pad_left,
                            float                scale,
                            float                mean0,
                            float                mean1,
                            float                mean2,
                            float                val0,
                            float                val1,
                            float                val2,
                            bool                 do_transpose,
                            bool                 do_norm,
                            int                  pad_value,
                            float                pad_color0,
                            float                pad_color1,
                            float                pad_color2,
                            hipStream_t         stream)
{
  dim3 block(16, 16);
  dim3 grid((dst_w + 15) / 16, (dst_h + 15) / 16);
  ResizePadNormPadvalueKernel<<<grid, block, 0, stream>>>(
      src, src_h, src_w, src_stride, src_format_bgr, dst, dst_h, dst_w, pad_top, pad_left, scale,
      mean0, mean1, mean2, val0, val1, val2, do_transpose, do_norm, pad_value, pad_color0,
      pad_color1, pad_color2);
}
}
